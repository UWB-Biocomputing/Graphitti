/**
 * @file AllIZHNeurons_d.cu
 *
 * @brief A container of all Izhikevich neuron data
 *
 * @ingroup Simulator/Vertices
 */

#include "AllSpikingSynapses.h"
#include "AllIZHNeurons.h"
#include "AllVerticesDeviceFuncs.h"

#include "Book.h"

///  Allocate GPU memories to store all neurons' states,
///  and copy them from host to GPU memory.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::allocNeuronDeviceStruct( void** allVerticesDevice ) {
	AllIZHNeuronsDeviceProperties allVerticesDeviceProps;

	allocDeviceStruct( allVerticesDeviceProps );

        HANDLE_ERROR( hipMalloc( allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allVerticesDevice, &allVerticesDeviceProps, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

///  Allocate GPU memories to store all neurons' states.
///  (Helper function of allocNeuronDeviceStruct)
///
///  @param  allVerticesDevice    GPU address of the AllIZHNeuronsDeviceProperties struct on device memory.
void AllIZHNeurons::allocDeviceStruct( AllIZHNeuronsDeviceProperties &allVerticesDevice ) {
	int count = Simulator::getInstance().getTotalVertices();

	AllIFNeurons::allocDeviceStruct( allVerticesDevice ); 
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Aconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Bconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Cconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Dconst_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.u_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.C3_, count * sizeof( BGFLOAT ) ) );
}

///  Delete GPU memories.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::deleteNeuronDeviceStruct( void* allVerticesDevice ) {
	AllIZHNeuronsDeviceProperties allVerticesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allVerticesDeviceProps );

	HANDLE_ERROR( hipFree( allVerticesDevice ) );
}

///  Delete GPU memories.
///  (Helper function of deleteNeuronDeviceStruct)
///
///  @param  allVerticesDevice    GPU address of the AllIZHNeuronsDeviceProperties struct on device memory.
void AllIZHNeurons::deleteDeviceStruct( AllIZHNeuronsDeviceProperties& allVerticesDevice ) {
	HANDLE_ERROR( hipFree( allVerticesDevice.Aconst_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Bconst_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Cconst_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Dconst_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.u_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.C3_ ) );

	AllIFNeurons::deleteDeviceStruct( allVerticesDevice );
}

///  Copy all neurons' data from host to device.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::copyNeuronHostToDevice( void* allVerticesDevice ) { 
	AllIZHNeuronsDeviceProperties allVerticesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allVerticesDeviceProps );
}

///  Copy all neurons' data from host to device.
///  (Helper function of copyNeuronHostToDevice)
///
///  @param  allVerticesDevice    GPU address of the AllIZHNeuronsDeviceProperties struct on device memory.
void AllIZHNeurons::copyHostToDevice( AllIZHNeuronsDeviceProperties& allVerticesDevice ) { 
	int count = Simulator::getInstance().getTotalVertices();

	AllIFNeurons::copyHostToDevice( allVerticesDevice );

	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Aconst_, &Aconst_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Bconst_, &Bconst_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Cconst_, &Cconst_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Dconst_, &Dconst_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.u_, &u_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.C3_, &C3_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
}

///  Copy all neurons' data from device to host.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::copyNeuronDeviceToHost( void* allVerticesDevice ) {
	AllIZHNeuronsDeviceProperties allVerticesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allVerticesDeviceProps );
}

///  Copy all neurons' data from device to host.
///  (Helper function of copyNeuronDeviceToHost)
///
///  @param  allVerticesDevice    GPU address of the AllIZHNeuronsDeviceProperties struct on device memory.
void AllIZHNeurons::copyDeviceToHost( AllIZHNeuronsDeviceProperties& allVerticesDevice ) {
	int count = Simulator::getInstance().getTotalVertices();

	AllIFNeurons::copyDeviceToHost( allVerticesDevice );

	HANDLE_ERROR( hipMemcpy ( &Aconst_[0], allVerticesDevice.Aconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Bconst_[0], allVerticesDevice.Bconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Cconst_[0], allVerticesDevice.Cconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Dconst_[0], allVerticesDevice.Dconst_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &u_[0], allVerticesDevice.u_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &C3_[0], allVerticesDevice.C3_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

///  Copy spike history data stored in device memory to host.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allVerticesDevice ) {
        AllIZHNeuronsDeviceProperties allVerticesDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allVerticesDeviceProps );
}

///  Copy spike counts data stored in device memory to host.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::copyNeuronDeviceSpikeCountsToHost( void* allVerticesDevice )
{
        AllIZHNeuronsDeviceProperties allVerticesDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allVerticesDeviceProps );
}

///  Clear the spike counts out of all neurons.
///
///  @param  allVerticesDevice   GPU address of the AllIZHNeuronsDeviceProperties struct 
///                             on device memory.
void AllIZHNeurons::clearNeuronSpikeCounts( void* allVerticesDevice )
{
        AllIZHNeuronsDeviceProperties allVerticesDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIZHNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allVerticesDeviceProps );
}

///  Notify outgoing synapses if neuron has fired.
void AllIZHNeurons::advanceVertices( AllEdges &synapses, void* allVerticesDevice, void* allEdgesDevice, float* randNoise, EdgeIndexMap* edgeIndexMapDevice)
{
    int vertex_count = Simulator::getInstance().getTotalVertices();
    int maxSpikes = (int)((Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate()));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( vertex_count + threadsPerBlock - 1 ) / threadsPerBlock;

    // Advance neurons ------------->
    advanceIZHNeuronsDevice <<< blocksPerGrid, threadsPerBlock >>> ( vertex_count, Simulator::getInstance().getMaxEdgesPerVertex(), maxSpikes, Simulator::getInstance().getDeltaT(), g_simulationStep, randNoise, (AllIZHNeuronsDeviceProperties *)allVerticesDevice, (AllSpikingSynapsesDeviceProperties*)allEdgesDevice, edgeIndexMapDevice, fAllowBackPropagation_ );
}
