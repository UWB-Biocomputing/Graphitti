/**
 * @file AllIFNeurons_d.cu
 *
 * @brief A container of all Integate and Fire (IF) neuron data
 *
 * @ingroup Simulator/Vertices
 */

#include "AllIFNeurons.h"
#include "Book.h"

///  Allocate GPU memories to store all neurons' states,
///  and copy them from host to GPU memory.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::allocNeuronDeviceStruct( void** allVerticesDevice ) {
	AllIFNeuronsDeviceProperties allNeurons;

	allocDeviceStruct( allNeurons );

        HANDLE_ERROR( hipMalloc( allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ) ) );
        HANDLE_ERROR( hipMemcpy ( *allVerticesDevice, &allNeurons, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyHostToDevice ) );
}

///  Allocate GPU memories to store all neurons' states.
///  (Helper function of allocNeuronDeviceStruct)
///
///  @param  allVerticesDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
void AllIFNeurons::allocDeviceStruct( AllIFNeuronsDeviceProperties &allVerticesDevice ) {
	int count = Simulator::getInstance().getTotalVertices();
	int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
 
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.C1_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.C2_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Cm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.I0_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Iinject_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Inoise_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Isyn_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Rm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Tau_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Trefract_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Vinit_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Vm_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Vreset_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Vrest_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.Vthresh_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.hasFired_, count * sizeof( bool ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.numStepsInRefractoryPeriod_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.spikeCount_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.spikeCountOffset_, count * sizeof( int ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.summationMap_, count * sizeof( BGFLOAT ) ) );
	HANDLE_ERROR( hipMalloc( ( void ** ) &allVerticesDevice.spikeHistory_, count * sizeof( uint64_t* ) ) );
	
	uint64_t* pSpikeHistory[count];
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipMalloc( ( void ** ) &pSpikeHistory[i], maxSpikes * sizeof( uint64_t ) ) );
	}
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.spikeHistory_, pSpikeHistory,
		count * sizeof( uint64_t* ), hipMemcpyHostToDevice ) );

	// get device summation point address
	summationMap_ = allVerticesDevice.summationMap_;
}

///  Delete GPU memories.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::deleteNeuronDeviceStruct( void* allVerticesDevice ) {
	AllIFNeuronsDeviceProperties allVerticesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allVerticesDeviceProps );

	HANDLE_ERROR( hipFree( allVerticesDevice ) );
}

///  Delete GPU memories.
///  (Helper function of deleteNeuronDeviceStruct)
///
///  @param  allVerticesDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
void AllIFNeurons::deleteDeviceStruct( AllIFNeuronsDeviceProperties& allVerticesDevice ) {
	int count = Simulator::getInstance().getTotalVertices();

	uint64_t* pSpikeHistory[count];
	HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allVerticesDevice.spikeHistory_,
		count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
	for (int i = 0; i < count; i++) {
		HANDLE_ERROR( hipFree( pSpikeHistory[i] ) );
	}

	HANDLE_ERROR( hipFree( allVerticesDevice.C1_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.C2_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Cm_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.I0_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Iinject_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Inoise_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Isyn_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Rm_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Tau_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Trefract_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Vinit_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Vm_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Vreset_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Vrest_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.Vthresh_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.hasFired_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.numStepsInRefractoryPeriod_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.spikeCount_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.spikeCountOffset_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.summationMap_ ) );
	HANDLE_ERROR( hipFree( allVerticesDevice.spikeHistory_ ) );
}

///  Copy all neurons' data from host to device.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::copyNeuronHostToDevice( void* allVerticesDevice ) { 
	AllIFNeuronsDeviceProperties allVerticesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyHostToDevice( allVerticesDeviceProps );
}

///  Copy all neurons' data from host to device.
///  (Helper function of copyNeuronHostToDevice)
///
///  @param  allVerticesDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
void AllIFNeurons::copyHostToDevice( AllIFNeuronsDeviceProperties& allVerticesDevice ) { 
	int count = Simulator::getInstance().getTotalVertices();

	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.C1_, &C1_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.C2_, &C2_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Cm_, &Cm_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.I0_, &I0_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Iinject_, &Iinject_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Inoise_, &Inoise_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Isyn_, &Isyn_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Rm_, &Rm_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Tau_, &Tau_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Trefract_, &Trefract_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Vinit_, &Vinit_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Vm_, &Vm_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Vreset_, &Vreset_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Vrest_, &Vrest_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.Vthresh_, &Vthresh_[0], count * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.hasFired_, &hasFired_[0], count * sizeof( bool ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.numStepsInRefractoryPeriod_, &numStepsInRefractoryPeriod_[0], count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.spikeCount_, &spikeCount_[0], count * sizeof( int ), hipMemcpyHostToDevice ) );
	HANDLE_ERROR( hipMemcpy ( allVerticesDevice.spikeCountOffset_, &spikeCountOffset_[0], count * sizeof( int ), hipMemcpyHostToDevice ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allVerticesDevice.spikeHistory_, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( pSpikeHistory[i], spikeHistory_[i], maxSpikes * sizeof( uint64_t ), hipMemcpyHostToDevice ) );
        }
}

///  Copy all neurons' data from device to host.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::copyNeuronDeviceToHost( void* allVerticesDevice ) {
	AllIFNeuronsDeviceProperties allVerticesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
	copyDeviceToHost( allVerticesDeviceProps );
}

///  Copy all neurons' data from device to host.
///  (Helper function of copyNeuronDeviceToHost)
///
///  @param  allVerticesDevice         GPU address of the AllIFNeuronsDeviceProperties struct.
void AllIFNeurons::copyDeviceToHost( AllIFNeuronsDeviceProperties& allVerticesDevice ) {
	int count = Simulator::getInstance().getTotalVertices();

	HANDLE_ERROR( hipMemcpy ( &C1_[0], allVerticesDevice.C1_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &C2_[0], allVerticesDevice.C2_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Cm_[0], allVerticesDevice.Cm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &I0_[0], allVerticesDevice.I0_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Iinject_[0], allVerticesDevice.Iinject_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Inoise_[0], allVerticesDevice.Inoise_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Isyn_[0], allVerticesDevice.Isyn_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Rm_[0], allVerticesDevice.Rm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Tau_[0], allVerticesDevice.Tau_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Trefract_[0], allVerticesDevice.Trefract_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Vinit_[0], allVerticesDevice.Vinit_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Vm_[0], allVerticesDevice.Vm_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Vreset_[0], allVerticesDevice.Vreset_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Vrest_[0], allVerticesDevice.Vrest_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &Vthresh_[0], allVerticesDevice.Vthresh_, count * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &hasFired_[0], allVerticesDevice.hasFired_, count * sizeof( bool ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &numStepsInRefractoryPeriod_[0], allVerticesDevice.numStepsInRefractoryPeriod_, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &spikeCount_[0], allVerticesDevice.spikeCount_, count * sizeof( int ), hipMemcpyDeviceToHost ) );
	HANDLE_ERROR( hipMemcpy ( &spikeCountOffset_[0], allVerticesDevice.spikeCountOffset_, count * sizeof( int ), hipMemcpyDeviceToHost ) );

        int maxSpikes = static_cast<int> (Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate());
        uint64_t* pSpikeHistory[count];
        HANDLE_ERROR( hipMemcpy ( pSpikeHistory, allVerticesDevice.spikeHistory_, count * sizeof( uint64_t* ), hipMemcpyDeviceToHost ) );
        for (int i = 0; i < count; i++) {
                HANDLE_ERROR( hipMemcpy ( spikeHistory_[i], pSpikeHistory[i], maxSpikes * sizeof( uint64_t ), hipMemcpyDeviceToHost ) );
        }
}

///  Copy spike history data stored in device memory to host.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::copyNeuronDeviceSpikeHistoryToHost( void* allVerticesDevice ) 
{        
        AllIFNeuronsDeviceProperties allVerticesDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );        
        AllSpikingNeurons::copyDeviceSpikeHistoryToHost( allVerticesDeviceProps );
}

///  Copy spike counts data stored in device memory to host.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::copyNeuronDeviceSpikeCountsToHost( void* allVerticesDevice )
{
        AllIFNeuronsDeviceProperties allVerticesDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::copyDeviceSpikeCountsToHost( allVerticesDeviceProps );
}

///  Clear the spike counts out of all neurons.
///
///  @param  allVerticesDevice   GPU address of the AllIFNeuronsDeviceProperties struct on device memory.
void AllIFNeurons::clearNeuronSpikeCounts( void* allVerticesDevice )
{
        AllIFNeuronsDeviceProperties allVerticesDeviceProps;
        HANDLE_ERROR( hipMemcpy ( &allVerticesDeviceProps, allVerticesDevice, sizeof( AllIFNeuronsDeviceProperties ), hipMemcpyDeviceToHost ) );
        AllSpikingNeurons::clearDeviceSpikeCounts( allVerticesDeviceProps );
}


///  Update the state of all neurons for a time step
///  Notify outgoing synapses if neuron has fired.
///
///  @param  synapses               Reference to the allEdges struct on host memory.
///  @param  allVerticesDevice       GPU address of the AllIFNeuronsDeviceProperties struct 
///                                 on device memory.
///  @param  allEdgesDevice      GPU address of the allEdgesDeviceProperties struct 
///                                 on device memory.
///  @param  randNoise              Reference to the random noise array.
///  @param  edgeIndexMapDevice  GPU address of the EdgeIndexMap on device memory.
void AllIFNeurons::advanceVertices( AllEdges &synapses, void* allVerticesDevice, void* allEdgesDevice, float* randNoise, EdgeIndexMap* edgeIndexMapDevice )
{
}
