#include "hip/hip_runtime.h"
/**
 * @file AsyncPhilox_d.cu
 * 
 * @ingroup Simulator/Utils/RNG
 * 
 * @brief Asynchronous Philox RNG using hiprand to fill GPU buffers
 * 
 * AsyncPhilox_d class maintains two large GPU buffers for noise.
 * GPUModel calls loadAsyncPhilox to initialize states and
 * fill the buffers, then, each advance requestSegment
 * returns a float* slice of a buffer for use in
 * advanceVertices
 */

#include "AsyncPhilox_d.h"
#include "NvtxHelper.h"
#include <cassert>
#include <chrono>
#include <iostream>

__global__ void generatePhilox(hiprandStatePhilox4_32_10_t *states, float *output, int bufferSize)
{
   // Compute a unique global index for this thread
   int threadId = threadIdx.x;
   int blockId = blockIdx.x;
   int threadsPerBlock = blockDim.x;
   int totalThreads = gridDim.x * threadsPerBlock;
   int gid = blockId * threadsPerBlock + threadId;

   // Load this thread’s Philox state
   hiprandStatePhilox4_32_10_t local = states[gid];

   // Stride‐loop: write one random per iteration until we cover bufferSize
   for (int idx = gid; idx < bufferSize; idx += totalThreads) {
      output[idx] = hiprand_normal(&local);
   }

   // Store back the updated state
   states[gid] = local;
}

__global__ void initPhilox(hiprandStatePhilox4_32_10_t *states, unsigned long seed, int totalThreads)
{
   int gid = blockIdx.x * blockDim.x + threadIdx.x;
   if (gid >= totalThreads)
      return;
   hiprand_init(seed, gid, 0, &states[gid]);
}

void AsyncPhilox_d::loadAsyncPhilox(int samplesPerSegment, unsigned long seed)
{
   // hostBuffer = nullptr;
   // hipHostAlloc(&hostBuffer, samplesPerSegment * sizeof(float), hipHostMallocDefault);
   // logfile = std::fopen("philox_output_32_10.bin", "wb");
   //consoleLogger_ = log4cplus::Logger::getInstance(LOG4CPLUS_TEXT("console"));
   segmentSize = samplesPerSegment;
   seed = seed;
   currentBuffer = 0;
   segmentIndex = 0;

   totalSegments = 10;

#ifdef ENABLE_NVTX
   nvtxMarker = 10000 / totalSegments;   // make a marker every nvtxMarker buffer fills;
   nvtxCurrentMarker = nvtxMarker;       // count down to color flip
#endif
   bufferSize = segmentSize * totalSegments;
   numBlocks = 64;   //placeholder num of blocks
   numThreads = 64;

   totalThreads = numThreads * numBlocks;

   int leastPriority, greatestPriority;
   HANDLE_ERROR(hipDeviceGetStreamPriorityRange(&leastPriority, &greatestPriority));
   // └─ leastPriority is the numerically largest value → lowest actual priority
   // └─ greatestPriority is the numerically smallest value → highest actual priority

   // Create internal stream
   HANDLE_ERROR(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, leastPriority));

   // Allocate two large buffers
   HANDLE_ERROR(hipMalloc(&buffers[0], bufferSize * sizeof(float)));
   HANDLE_ERROR(hipMalloc(&buffers[1], bufferSize * sizeof(float)));

   HANDLE_ERROR(hipMalloc(&spStates, totalThreads * sizeof(hiprandStatePhilox4_32_10_t)));

   initPhilox<<<totalThreads + 255 / 256, 256, 0, stream>>>(spStates, seed, totalThreads);

   // Pre-fill both buffers
   fillBuffer(0);
   fillBuffer(1);
   HANDLE_ERROR(hipStreamSynchronize(
      stream));   //wait for both buffers to be filled before the first request
}
void AsyncPhilox_d::deleteDeviceStruct()
{
   // std::fclose(logfile);
   // hipFree(hostBuffer);
   HANDLE_ERROR(hipFree(buffers[0]));
   HANDLE_ERROR(hipFree(buffers[1]));
   HANDLE_ERROR(hipFree(spStates));

   HANDLE_ERROR(hipStreamDestroy(stream));
}
AsyncPhilox_d::~AsyncPhilox_d()
{
}

float *AsyncPhilox_d::requestSegment()
{
   //LOG4CPLUS_TRACE(consoleLogger_, "request segment");
   //auto start = std::chrono::high_resolution_clock::now();
   static bool flipColor;
   if (segmentIndex >= totalSegments) {
      // Switch buffer and launch async refill on the now-unused one

#ifdef ENABLE_NVTX
      if (nvtxCurrentMarker <= 0) {
         nvtxPop();
         if (flipColor == true)
            nvtxPushColor("10,000 time steps", Color::RED);
         else
            nvtxPushColor("10,000 time steps", Color::BLUE);

         flipColor = !flipColor;
         nvtxCurrentMarker = nvtxMarker;
      } else
         --nvtxCurrentMarker;
#endif

      int refillBuffer = currentBuffer;
      currentBuffer = 1 - currentBuffer;
      segmentIndex = 0;
      hipStreamSynchronize(stream);   // Ensure refillBuffer is done
      fillBuffer(refillBuffer);
   }

   float *segmentPtr = buffers[currentBuffer] + segmentIndex * segmentSize;
   segmentIndex += 1;

   // auto end = std::chrono::high_resolution_clock::now();
   // std::cout << "Segment: " << segmentIndex << ", Launch time: " << (end - start).count() << " ns\n";
   // hipMemcpy(hostBuffer, segmentPtr, segmentSize * sizeof(float), hipMemcpyDeviceToHost);
   // std::fwrite(hostBuffer, sizeof(float), segmentSize, logfile);

   return segmentPtr;
}

void AsyncPhilox_d::fillBuffer(int bufferIndex)
{
   //LOG4CPLUS_TRACE(consoleLogger_, "filling buffer:");
   generatePhilox<<<numBlocks, numThreads, 0, stream>>>(spStates, buffers[bufferIndex], bufferSize);
}
