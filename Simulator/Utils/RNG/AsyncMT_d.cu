#include "hip/hip_runtime.h"
#include "AsyncMT_d.h"
#include <cassert>
#include <rocrand/rocrand_mtgp32_11213.h>
#include <iostream>
#include <chrono>
__global__ void generateKernel(hiprandStateMtgp32 *state, float *output, int samplesPerGen)
{
   int tid = threadIdx.x;
   int gen_id = blockIdx.x;
   if (gen_id >= gridDim.x)
      return;

   hiprandStateMtgp32 localState = state[gen_id];
   for (int i = tid; i < samplesPerGen; i += blockDim.x) {
      output[gen_id * samplesPerGen + i] = hiprand_normal(&localState);
   }
   state[gen_id] = localState;
}

void AsyncMT_d::loadAsyncMT(int samplesPerSegment, unsigned long seed)
{
   // hostBuffer = nullptr;
   // hipHostAlloc(&hostBuffer, samplesPerSegment * sizeof(float), hipHostMallocDefault);
   // logfile = std::fopen("mt_output.bin", "wb");
   //consoleLogger_ = log4cplus::Logger::getInstance(LOG4CPLUS_TEXT("console"));
   segmentSize = samplesPerSegment;
   seed = seed;
   currentBuffer = 0;
   segmentIndex = 0;
   totalSegments = 10000;   // Each buffer has 10000 segments
   bufferSize = segmentSize * totalSegments;
   totalSamples = bufferSize * 2;
   numGenerators = 50;   //placeholder num of blocks

   // Create internal stream
   HANDLE_ERROR(hipStreamCreate(&stream));

   // Allocate two large buffers
   HANDLE_ERROR(hipMalloc(&buffers[0], bufferSize * sizeof(float)));
   HANDLE_ERROR(hipMalloc(&buffers[1], bufferSize * sizeof(float)));

   // Allocate state and param memory
   HANDLE_ERROR(hipMalloc(&d_states, numGenerators * sizeof(hiprandStateMtgp32)));
   HANDLE_ERROR(hipMalloc(&d_params, numGenerators * sizeof(mtgp32_kernel_params_t)));


   // Create local param buffer of correct type
   mtgp32_kernel_params_t *h_params = new mtgp32_kernel_params_t[numGenerators];
   hiprandMakeMTGP32Constants(mtgp32dc_params_fast_11213, h_params);
   HANDLE_ERROR(hipMemcpy(d_params, h_params, numGenerators * sizeof(mtgp32_kernel_params_t),
              hipMemcpyHostToDevice));
   delete[] h_params;

   hiprandMakeMTGP32KernelState(d_states, mtgp32dc_params_fast_11213, d_params, numGenerators, seed);

   // Pre-fill both buffers
   fillBuffer(0);
   fillBuffer(1);
   HANDLE_ERROR(hipStreamSynchronize(stream)); //wait for both buffers to be filled before the first request
}
void AsyncMT_d::deleteDeviceStruct(){
   // std::fclose(logfile);
   // hipFree(hostBuffer);
   HANDLE_ERROR(hipFree(buffers[0]));
   HANDLE_ERROR(hipFree(buffers[1]));
   HANDLE_ERROR(hipFree(d_states));
   HANDLE_ERROR(hipFree(d_params));
   HANDLE_ERROR(hipStreamDestroy(stream));
}
AsyncMT_d::~AsyncMT_d()
{
}

float *AsyncMT_d::requestSegment()
{
   //LOG4CPLUS_TRACE(consoleLogger_, "request segment");
   //auto start = std::chrono::high_resolution_clock::now();
   if (segmentIndex >= totalSegments) {
      // Switch buffer and launch async refill on the now-unused one
      int refillBuffer = currentBuffer;
      currentBuffer = 1 - currentBuffer;
      segmentIndex = 0;
      hipStreamSynchronize(stream);   // Ensure refillBuffer is done
      fillBuffer(refillBuffer);
      //hipStreamSynchronize(stream);
   }

   float *segmentPtr = buffers[currentBuffer] + segmentIndex * segmentSize;
   segmentIndex += 1;

   // auto end = std::chrono::high_resolution_clock::now();
   // std::cout << "Segment: " << segmentIndex << ", Launch time: " << (end - start).count() << " ns\n";
   // hipMemcpy(hostBuffer, segmentPtr, segmentSize * sizeof(float), hipMemcpyDeviceToHost);
   // std::fwrite(hostBuffer, sizeof(float), segmentSize, logfile);
   return segmentPtr;
}

void AsyncMT_d::fillBuffer(int bufferIndex)
{
   dim3 blocks(numGenerators);
   dim3 threads(256);
   //LOG4CPLUS_TRACE(consoleLogger_, "filling buffer:");
   generateKernel<<<blocks, threads, 0, stream>>>(d_states, buffers[bufferIndex],
                                                  bufferSize / numGenerators);
}
