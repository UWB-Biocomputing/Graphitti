/**
 * @file ConnGrowth_d.cu
 * 
 * @ingroup Simulator/Connections
 *
 * @brief Update the weights of the Synapses in the simulation.
 */

#include "ConnGrowth.h"
#include "AllSpikingSynapses.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Simulator.h"
#include "Book.h"

/*
 *  Update the weights of the Synapses in the simulation. To be clear,
 *  iterates through all source and destination vertices and updates their
 *  edge strengths from the weight matrix.
 *  Note: Platform Dependent.
 *
 *  @param  numVertices         number of vertices to update.
 *  @param  vertices            The AllVertices object.
 *  @param  synapses           The AllEdges object.
 *  @param  allVerticesDevice   GPU address to the AllVertices struct in device memory.
 *  @param  allEdgesDevice  GPU address to the allEdges struct in device memory.
 *  @param  layout             The Layout object.
 */
void ConnGrowth::updateSynapsesWeights(const int numVertices, AllVertices &vertices, AllEdges &synapses, AllSpikingNeuronsDeviceProperties* allVerticesDevice, AllSpikingSynapsesDeviceProperties* allEdgesDevice, Layout *layout)
{
        Simulator &simulator = Simulator::getInstance();
        // For now, we just set the weights to equal the areas. We will later
        // scale it and set its sign (when we index and get its sign).
        (*W_) = (*area_);

        BGFLOAT deltaT = simulator.getDeltaT();

        // CUDA parameters
        const int threadsPerBlock = 256;
        int blocksPerGrid;

        // allocate device memories
        BGSIZE W_d_size = simulator.getTotalVertices() * simulator.getTotalVertices() * sizeof (BGFLOAT);
        BGFLOAT* W_h = new BGFLOAT[W_d_size];
        BGFLOAT* W_d;
        HANDLE_ERROR( hipMalloc ( ( void ** ) &W_d, W_d_size ) );

        vertexType* neuronTypeMapD;
        HANDLE_ERROR( hipMalloc( ( void ** ) &neuronTypeMapD, simulator.getTotalVertices() * sizeof( vertexType ) ) );

        // copy weight data to the device memory
        for ( int i = 0 ; i < simulator.getTotalVertices(); i++ )
                for ( int j = 0; j < simulator.getTotalVertices(); j++ )
                        W_h[i * simulator.getTotalVertices() + j] = (*W_)(i, j);

        HANDLE_ERROR( hipMemcpy ( W_d, W_h, W_d_size, hipMemcpyHostToDevice ) );

        HANDLE_ERROR( hipMemcpy ( neuronTypeMapD, layout->vertexTypeMap_, simulator.getTotalVertices() * sizeof( vertexType ), hipMemcpyHostToDevice ) );

        blocksPerGrid = ( simulator.getTotalVertices() + threadsPerBlock - 1 ) / threadsPerBlock;
        updateSynapsesWeightsDevice <<< blocksPerGrid, threadsPerBlock >>> ( simulator.getTotalVertices(), deltaT, W_d, simulator.getMaxEdgesPerVertex(), allVerticesDevice, allEdgesDevice, neuronTypeMapD );

        // free memories
        HANDLE_ERROR( hipFree( W_d ) );
        delete[] W_h;

        HANDLE_ERROR( hipFree( neuronTypeMapD ) );

        // copy device synapse count to host memory
        synapses.copyDeviceEdgeCountsToHost(allEdgesDevice);
        // copy device synapse summation coordinate to host memory
        synapses.copyDeviceEdgeSumIdxToHost(allEdgesDevice);
}
