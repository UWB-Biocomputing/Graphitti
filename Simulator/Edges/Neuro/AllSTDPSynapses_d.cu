/**
 * @file AllSTDPSynapses_d.cu
 * 
 * @ingroup Simulator/Edges
 *
 * @brief A container of all STDP synapse data
 */

#include "AllSTDPSynapses.h"
#include "AllSpikingSynapses.h"
#include "GPUModel.h"
#include "AllSynapsesDeviceFuncs.h"
#include "Book.h"

///  Allocate GPU memories to store all synapses' states,
///  and copy them from host to GPU memory.
///
///  @param  allEdgesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                             on device memory.
void AllSTDPSynapses::allocEdgeDeviceStruct( void** allEdgesDevice ) {
	allocEdgeDeviceStruct( allEdgesDevice, Simulator::getInstance().getTotalVertices(), Simulator::getInstance().getMaxEdgesPerVertex() );
}

///  Allocate GPU memories to store all synapses' states,
///  and copy them from host to GPU memory.
///
///  @param  allEdgesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                                on device memory.
///  @param  numVertices            Number of vertices.
///  @param  maxEdgesPerVertex  Maximum number of synapses per neuron.
void AllSTDPSynapses::allocEdgeDeviceStruct( void** allEdgesDevice, int numVertices, int maxEdgesPerVertex ) {
	AllSTDPSynapsesDeviceProperties allEdgesDeviceProps;

	allocDeviceStruct( allEdgesDeviceProps, numVertices, maxEdgesPerVertex );

	HANDLE_ERROR( hipMalloc( allEdgesDevice, sizeof( AllSTDPSynapsesDeviceProperties ) ) );
	HANDLE_ERROR( hipMemcpy ( *allEdgesDevice, &allEdgesDeviceProps, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyHostToDevice ) );
}

///  Allocate GPU memories to store all synapses' states,
///  and copy them from host to GPU memory.
///  (Helper function of allocEdgeDeviceStruct)
///
///  @param  allEdgesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                                on device memory.
///  @param  numVertices            Number of vertices.
///  @param  maxEdgesPerVertex  Maximum number of synapses per neuron.
void AllSTDPSynapses::allocDeviceStruct( AllSTDPSynapsesDeviceProperties &allEdgesDevice, int numVertices, int maxEdgesPerVertex ) {
        AllSpikingSynapses::allocDeviceStruct( allEdgesDevice, numVertices, maxEdgesPerVertex );

        BGSIZE maxTotalSynapses = maxEdgesPerVertex * numVertices;

        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.totalDelayPost_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.delayQueuePost_, maxTotalSynapses * sizeof( BGSIZE ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.delayIndexPost_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.delayQueuePost_, maxTotalSynapses * sizeof( int ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.tauspost_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.tauspre_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.taupos_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.tauneg_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.STDPgap_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.Wex_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.Aneg_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.Apos_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.mupos_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
        HANDLE_ERROR( hipMalloc( ( void ** ) &allEdgesDevice.muneg_, maxTotalSynapses * sizeof( BGFLOAT ) ) );
}

///  Delete GPU memories.
///
///  @param  allEdgesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                             on device memory.
void AllSTDPSynapses::deleteEdgeDeviceStruct( void* allEdgesDevice ) {
	AllSTDPSynapsesDeviceProperties allEdgesDeviceProps;

	HANDLE_ERROR( hipMemcpy ( &allEdgesDeviceProps, allEdgesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	deleteDeviceStruct( allEdgesDeviceProps );

	HANDLE_ERROR( hipFree( allEdgesDevice ) );
}

///  Delete GPU memories.
///  (Helper function of deleteEdgeDeviceStruct)
///
///  @param  allEdgesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                             on device memory.
void AllSTDPSynapses::deleteDeviceStruct( AllSTDPSynapsesDeviceProperties& allEdgesDevice ) {
        HANDLE_ERROR( hipFree( allEdgesDevice.totalDelayPost_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.delayQueuePost_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.delayIndexPost_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.tauspost_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.tauspre_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.taupos_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.tauneg_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.STDPgap_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.Wex_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.Aneg_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.Apos_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.mupos_ ) );
        HANDLE_ERROR( hipFree( allEdgesDevice.muneg_) );

        AllSpikingSynapses::deleteDeviceStruct( allEdgesDevice );
}

///  Copy all synapses' data from host to device.
///
///  @param  allEdgesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                                on device memory.
///  @param  numVertices            Number of vertices.
///  @param  maxEdgesPerVertex  Maximum number of synapses per neuron.
void AllSTDPSynapses::copyEdgeHostToDevice( void* allEdgesDevice ) { // copy everything necessary
	copyEdgeHostToDevice( allEdgesDevice, Simulator::getInstance().getTotalVertices(), Simulator::getInstance().getMaxEdgesPerVertex() );	
}

///  Copy all synapses' data from host to device.
///
///  @param  allEdgesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                                on device memory.
///  @param  numVertices            Number of vertices.
///  @param  maxEdgesPerVertex  Maximum number of synapses per neuron.
void AllSTDPSynapses::copyEdgeHostToDevice( void* allEdgesDevice, int numVertices, int maxEdgesPerVertex ) { // copy everything necessary
	AllSTDPSynapsesDeviceProperties allEdgesDeviceProps;

        HANDLE_ERROR( hipMemcpy ( &allEdgesDeviceProps, allEdgesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyHostToDevice( allEdgesDevice, allEdgesDeviceProps, numVertices, maxEdgesPerVertex );	
}

///  Copy all synapses' data from host to device.
///  (Helper function of copyEdgeHostToDevice)
///
///  @param  allEdgesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                                on device memory.
///  @param  numVertices            Number of vertices.
///  @param  maxEdgesPerVertex  Maximum number of synapses per neuron.
void AllSTDPSynapses::copyHostToDevice( void* allEdgesDevice, AllSTDPSynapsesDeviceProperties& allEdgesDeviceProps, int numVertices, int maxEdgesPerVertex ) { // copy everything necessary 
        AllSpikingSynapses::copyHostToDevice( allEdgesDevice, allEdgesDeviceProps, numVertices, maxEdgesPerVertex );

        BGSIZE maxTotalSynapses = maxEdgesPerVertex * numVertices;
        
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.totalDelayPost_, &totalDelayPost_[0],
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.delayQueuePost_, &delayQueuePost_[0],
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.delayIndexPost_, &delayIndexPost_[0],
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.delayQueuePost_, &delayQueuePost_[0],
                maxTotalSynapses * sizeof( int ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.tauspost_, &tauspost_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.tauspre_, &tauspre_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.taupos_, &taupos_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.tauneg_, &tauneg_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.STDPgap_, &STDPgap_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.Wex_, &Wex_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.Aneg_, &Aneg_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.Apos_, &Apos_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.mupos_, &mupos_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
        HANDLE_ERROR( hipMemcpy ( allEdgesDeviceProps.muneg_, &muneg_[0],
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyHostToDevice ) ); 
}

///  Copy all synapses' data from device to host.
///
///  @param  allEdgesDevice  GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                             on device memory.
void AllSTDPSynapses::copyEdgeDeviceToHost( void* allEdgesDevice ) {
	// copy everything necessary
	AllSTDPSynapsesDeviceProperties allEdgesDeviceProps;

        HANDLE_ERROR( hipMemcpy ( &allEdgesDeviceProps, allEdgesDevice, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );

	copyDeviceToHost( allEdgesDeviceProps );
}

///  Copy all synapses' data from device to host.
///  (Helper function of copyEdgeDeviceToHost)
///
///  @param  allEdgesDevice     GPU address of the AllSTDPSynapsesDeviceProperties struct 
///                                on device memory.
///  @param  numVertices            Number of vertices.
///  @param  maxEdgesPerVertex  Maximum number of synapses per neuron.
void AllSTDPSynapses::copyDeviceToHost( AllSTDPSynapsesDeviceProperties& allEdgesDevice ) {
        AllSpikingSynapses::copyDeviceToHost( allEdgesDevice ) ;

	int numVertices = Simulator::getInstance().getTotalVertices();
	BGSIZE maxTotalSynapses = Simulator::getInstance().getMaxEdgesPerVertex() * numVertices;

        HANDLE_ERROR( hipMemcpy ( &delayQueuePost_[0], allEdgesDevice.delayQueuePost_,
                maxTotalSynapses * sizeof( uint32_t ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &delayIndexPost_[0], allEdgesDevice.delayIndexPost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &delayQueuePost_[0], allEdgesDevice.delayQueuePost_,
                maxTotalSynapses * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &tauspost_[0], allEdgesDevice.tauspost_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &tauspre_[0], allEdgesDevice.tauspre_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &taupos_[0], allEdgesDevice.taupos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &tauneg_[0], allEdgesDevice.tauneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &STDPgap_[0], allEdgesDevice.STDPgap_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &Wex_[0], allEdgesDevice.Wex_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &Aneg_[0], allEdgesDevice.Aneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &Apos_[0], allEdgesDevice.Apos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &mupos_[0], allEdgesDevice.mupos_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( &muneg_[0], allEdgesDevice.muneg_,
                maxTotalSynapses * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
}

///  Advance all the Synapses in the simulation.
///  Update the state of all synapses for a time step.
///
///  @param  allEdgesDevice      GPU address of the AllEdgesDeviceProperties struct 
///                                 on device memory.
///  @param  allVerticesDevice       GPU address of the allNeurons struct on device memory.
///  @param  edgeIndexMapDevice  GPU address of the EdgeIndexMap on device memory.
void AllSTDPSynapses::advanceEdges( void* allEdgesDevice, void* allVerticesDevice, void* edgeIndexMapDevice )
{
    int maxSpikes = (int) ((Simulator::getInstance().getEpochDuration() * Simulator::getInstance().getMaxFiringRate()));

    // CUDA parameters
    const int threadsPerBlock = 256;
    int blocksPerGrid = ( totalEdgeCount_ + threadsPerBlock - 1 ) / threadsPerBlock;
    // Advance synapses ------------->
    advanceSTDPSynapsesDevice <<< blocksPerGrid, threadsPerBlock >>> ( totalEdgeCount_, (EdgeIndexMap*) edgeIndexMapDevice, g_simulationStep, Simulator::getInstance().getDeltaT(), 
                                (AllSTDPSynapsesDeviceProperties*)allEdgesDevice, (AllSpikingNeuronsDeviceProperties*)allVerticesDevice, maxSpikes );
}
    
///  Set synapse class ID defined by enumClassSynapses for the caller's Synapse class.
///  The class ID will be set to classSynapses_d in device memory,
///  and the classSynapses_d will be referred to call a device function for the
///  particular synapse class.
///  Because we cannot use virtual function (Polymorphism) in device functions,
///  we use this scheme.
///  Note: we used to use a function pointer; however, it caused the growth_cuda crash
///  (see issue#137).
void AllSTDPSynapses::setEdgeClassID()
{
    enumClassSynapses classSynapses_h = classAllSTDPSynapses;

    HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(classSynapses_d), &classSynapses_h, sizeof(enumClassSynapses)) );
}

///  Prints GPU SynapsesProps data.
///   
///  @param  allEdgesDeviceProps   GPU address of the corresponding SynapsesDeviceProperties struct on device memory.
void AllSTDPSynapses::printGPUEdgesProps( void* allEdgesDeviceProps ) const
{
    AllSTDPSynapsesDeviceProperties allSynapsesProps;

    //allocate print out data members
    BGSIZE size = maxEdgesPerVertex_ * countVertices_;
    if (size != 0) {
        BGSIZE *synapseCountsPrint = new BGSIZE[countVertices_];
        BGSIZE maxEdgesPerVertexPrint;
        BGSIZE totalSynapseCountPrint;
        int countNeuronsPrint;
        int *sourceNeuronIndexPrint = new int[size];
        int *destNeuronIndexPrint = new int[size];
        BGFLOAT *WPrint = new BGFLOAT[size];

        edgeType *typePrint = new edgeType[size];
        BGFLOAT *psrPrint = new BGFLOAT[size];
        bool *inUsePrint = new bool[size];

        for (BGSIZE i = 0; i < size; i++) {
            inUsePrint[i] = false;
        }

        for (int i = 0; i < countVertices_; i++) {
            synapseCountsPrint[i] = 0;
        }

        BGFLOAT *decayPrint = new BGFLOAT[size];
        int *totalDelayPrint = new int[size];
        BGFLOAT *tauPrint = new BGFLOAT[size];

        int *totalDelayPostPrint = new int[size];
        BGFLOAT *tauspostPrint = new BGFLOAT[size];
        BGFLOAT *tausprePrint = new BGFLOAT[size];
        BGFLOAT *tauposPrint = new BGFLOAT[size];
        BGFLOAT *taunegPrint = new BGFLOAT[size];
        BGFLOAT *STDPgapPrint = new BGFLOAT[size];
        BGFLOAT *WexPrint = new BGFLOAT[size];
        BGFLOAT *AnegPrint = new BGFLOAT[size];
        BGFLOAT *AposPrint = new BGFLOAT[size];
        BGFLOAT *muposPrint = new BGFLOAT[size];
        BGFLOAT *munegPrint = new BGFLOAT[size];

        // copy everything
        HANDLE_ERROR( hipMemcpy ( &allSynapsesProps, allEdgesDeviceProps, sizeof( AllSTDPSynapsesDeviceProperties ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( synapseCountsPrint, allSynapsesProps.edgeCounts_, countVertices_ * sizeof( BGSIZE ), hipMemcpyDeviceToHost ) );
        maxEdgesPerVertexPrint = allSynapsesProps.maxEdgesPerVertex_;
        totalSynapseCountPrint = allSynapsesProps.totalEdgeCount_;
        countNeuronsPrint = allSynapsesProps.countVertices_;

        // Set countVertices_ to 0 to avoid illegal memory deallocation
        // at AllSynapsesProps deconstructor.
        allSynapsesProps.countVertices_ = 0;

        HANDLE_ERROR( hipMemcpy ( sourceNeuronIndexPrint, allSynapsesProps.sourceVertexIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( destNeuronIndexPrint, allSynapsesProps.destVertexIndex_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WPrint, allSynapsesProps.W_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( typePrint, allSynapsesProps.type_, size * sizeof( edgeType ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( psrPrint, allSynapsesProps.psr_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( inUsePrint, allSynapsesProps.inUse_, size * sizeof( bool ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( decayPrint, allSynapsesProps.decay_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauPrint, allSynapsesProps.tau_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( totalDelayPrint, allSynapsesProps.totalDelay_, size * sizeof( int ), hipMemcpyDeviceToHost ) );

        HANDLE_ERROR( hipMemcpy ( totalDelayPostPrint, allSynapsesProps.totalDelayPost_, size * sizeof( int ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauspostPrint, allSynapsesProps.tauspost_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tausprePrint, allSynapsesProps.tauspre_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( tauposPrint, allSynapsesProps.taupos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( taunegPrint, allSynapsesProps.tauneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( STDPgapPrint, allSynapsesProps.STDPgap_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( WexPrint, allSynapsesProps.Wex_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( AnegPrint, allSynapsesProps.Aneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( AposPrint, allSynapsesProps.Apos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( muposPrint, allSynapsesProps.mupos_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );
        HANDLE_ERROR( hipMemcpy ( munegPrint, allSynapsesProps.muneg_, size * sizeof( BGFLOAT ), hipMemcpyDeviceToHost ) );

        for(int i = 0; i < maxEdgesPerVertex_ * countVertices_; i++) {
            if (WPrint[i] != 0.0) {
                cout << "GPU W[" << i << "] = " << WPrint[i];
                cout << " GPU sourNeuron: " << sourceNeuronIndexPrint[i];
                cout << " GPU desNeuron: " << destNeuronIndexPrint[i];
                cout << " GPU type: " << typePrint[i];
                cout << " GPU psr: " << psrPrint[i];
                cout << " GPU in_use:" << inUsePrint[i];

                cout << " GPU decay: " << decayPrint[i];
                cout << " GPU tau: " << tauPrint[i];
                cout << " GPU total_delay: " << totalDelayPrint[i];

                cout << " GPU total_delayPost: " << totalDelayPostPrint[i];
                cout << " GPU tauspost_: " << tauspostPrint[i];
                cout << " GPU tauspre_: " << tausprePrint[i];
                cout << " GPU taupos_: " << tauposPrint[i];
                cout << " GPU tauneg_: " << taunegPrint[i];
                cout << " GPU STDPgap_: " << STDPgapPrint[i];
                cout << " GPU Wex_: " << WexPrint[i];
                cout << " GPU Aneg_: " << AnegPrint[i];
                cout << " GPU Apos_: " << AposPrint[i];
                cout << " GPU mupos_: " << muposPrint[i];
                cout << " GPU muneg_: " << munegPrint[i] << endl;
            }
        }

        for (int i = 0; i < countVertices_; i++) {
            cout << "GPU edge_counts:" << "neuron[" << i  << "]" << synapseCountsPrint[i] << endl;
        }

        cout << "GPU totalSynapseCount:" << totalSynapseCountPrint << endl;
        cout << "GPU maxEdgesPerVertex:" << maxEdgesPerVertexPrint << endl;
        cout << "GPU countVertices_:" << countNeuronsPrint << endl;

        // Set countVertices_ to 0 to avoid illegal memory deallocation
        // at AllDSSynapsesProps deconstructor.
        allSynapsesProps.countVertices_ = 0;

        delete[] destNeuronIndexPrint;
        delete[] WPrint;
        delete[] sourceNeuronIndexPrint;
        delete[] psrPrint;
        delete[] typePrint;
        delete[] inUsePrint;
        delete[] synapseCountsPrint;
        destNeuronIndexPrint = nullptr;
        WPrint = nullptr;
        sourceNeuronIndexPrint = nullptr;
        psrPrint = nullptr;
        typePrint = nullptr;
        inUsePrint = nullptr;
        synapseCountsPrint = nullptr;

        delete[] decayPrint;
        delete[] totalDelayPrint;
        delete[] tauPrint;
        decayPrint = nullptr;
        totalDelayPrint = nullptr;
        tauPrint = nullptr;

        delete[] totalDelayPostPrint;
        delete[] tauspostPrint;
        delete[] tausprePrint;
        delete[] tauposPrint;
        delete[] taunegPrint;
        delete[] STDPgapPrint;
        delete[] WexPrint;
        delete[] AnegPrint;
        delete[] AposPrint;
        delete[] muposPrint;
        delete[] munegPrint;
        totalDelayPostPrint = nullptr;
        tauspostPrint = nullptr;
        tausprePrint = nullptr;
        tauposPrint = nullptr;
        taunegPrint = nullptr;
        STDPgapPrint = nullptr;
        WexPrint = nullptr;
        AnegPrint = nullptr;
        AposPrint = nullptr;
        muposPrint = nullptr;
        munegPrint = nullptr;
    }

}
